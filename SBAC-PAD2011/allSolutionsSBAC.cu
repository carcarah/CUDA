#include "hip/hip_runtime.h"

/*
 * main.c
 *
 *  Created on: 26/01/2011
 *      Author: einstein/carneiro
 */
#include <stdio.h>
#include <string.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

#define mat_h(i,j) mat_h[i*N+j]
#define mat_d(i,j) mat_d[i*N_l+j]
#define mat_block(i,j) mat_block[i*N_l+j]
#define proximo(x) x+1
#define anterior(x) x-1
#define MAX 8192
#define INFINITO 999999
#define ZERO 0
#define ONE 1

#define _VAZIO_      -1
#define _VISITADO_    1
#define _NAO_VISITADO_ 0

int qtd = 0;
int custo = 0;
int N;
int melhor = INFINITO;
int upper_bound;

int mat_h[MAX];


static void HandleError( hipError_t err,
                         const char *file,
                         int line ) {
    if (err != hipSuccess) {
        printf( "%s in %s at line %d\n", hipGetErrorString( err ),
                file, line );
        exit( EXIT_FAILURE );
    }
}
#define HANDLE_ERROR( err ) (HandleError( err, __FILE__, __LINE__ ))


#define HANDLE_NULL( a ) {if (a == NULL) { \
                            printf( "Host memory failed in %s at line %d\n", \
                                    __FILE__, __LINE__ ); \
                            exit( EXIT_FAILURE );}}

void read() {
	int i;
	//scanf("%d", &upper_bound);
	scanf("%d", &N);
	for (i = 0; i < (N * N); i++) {
		scanf("%d", &mat_h[i]);
	}

}

int calculaNPrefixos(int nivelPrefixo, int nVertice) {
	int x = nVertice - 1;
	int i;
	for (i = 1; i < nivelPrefixo-1; ++i) {
		x *= nVertice - i-1;
	}
	return x;
}

void fillFixedPaths(short* preFixo, int nivelPrefixo) {
	char flag[16];
	int vertice[16]; //representa o ciclo
	int cont = 0;
	int i, nivel; //para dizer que 0-1 ja foi visitado e a busca comeca de 1, bote 2


	for (i = 0; i < N; ++i) {
		flag[i] = 0;
		vertice[i] = -1;
	}

	vertice[0] = 0; //aqui!!!! vertice[nivel] = idx vflag[idx] = 1
	flag[0] = 1;
	nivel = 1;
	while (nivel >= 1) { // modificar aqui se quiser comecar a busca de determinado nivel

		if (vertice[nivel] != -1) {
			flag[vertice[nivel]] = 0;
		}

		do {
			vertice[nivel]++;
		} while (vertice[nivel] < N && flag[vertice[nivel]]); //




		if (vertice[nivel] < N) { //vertice[x] vertice no nivel x


			flag[vertice[nivel]] = 1;
			nivel++;

			if (nivel == nivelPrefixo) {
				for (i = 0; i < nivelPrefixo; ++i) {
					preFixo[cont * nivelPrefixo + i] = vertice[i];
//					printf("%d ", vertice[i]);
				}
//				printf("\n");
				cont++;
				nivel--;
			}
		} else {
			vertice[nivel] = -1;
			nivel--;
		}//else
	}//while
}


/*@OK: ter N e UB locais.
	
  @TODO em 7: mat_d por bloco? õ0 Prefixos compartilhados? õ0

*/
__global__ void dfs_cuda_UB(int N, int *mat_d, short *preFixos_d,
		int nPreFixos, int nivelPrefixo, int upper_bound, int *sols_d,int *melhorSol_d) {

	register int idx = blockIdx.x * blockDim.x + threadIdx.x;
	register int flag[16];
	register int vertice[16]; //representa o ciclo
	
	register int N_l = N;
	
	register int i, nivel; //para dizer que 0-1 ja foi visitado e a busca comeca de 1, bote 2
	register int custo;
	register int qtd_solucoes_thread = 0;
	register int UB_local = upper_bound;
	register int nivelGlobal = nivelPrefixo;

	if (idx < nPreFixos) { //(@)botar algo com vflag aqui, pois do jeito que esta algumas threads tentarao descer.
			
		for (i = 0; i < N_l; ++i) {
			vertice[i] = _VAZIO_;
			flag[i] = _NAO_VISITADO_;
		}
		
		vertice[0] = 0;
		flag[0] = _VISITADO_;
		custo= ZERO;
		
		for (i = 1; i < nivelGlobal; ++i) {
			vertice[i] = preFixos_d[idx * nivelGlobal + i];
			flag[vertice[i]] = _VISITADO_;
			custo += mat_d(vertice[i-1],vertice[i]);
		}
		
		nivel=nivelPrefixo;

	

		while (nivel >= nivelGlobal ) { // modificar aqui se quiser comecar a busca de determinado nivel

			if (vertice[nivel] != _VAZIO_) {
				flag[vertice[nivel]] = _NAO_VISITADO_;
				custo -= mat_d(vertice[anterior(nivel)],vertice[nivel]);
			}

			do {
				vertice[nivel]++;
			} while (vertice[nivel] < N_l && flag[vertice[nivel]]); //


			if (vertice[nivel] < N_l) { //vertice[x] vertice no nivel x
				custo += mat_d(vertice[anterior(nivel)],vertice[nivel]);
				flag[vertice[nivel]] = _VISITADO_;
				nivel++;

				if (nivel == N_l) { //se o vértice do nível for == N, entao formou o ciclo e vc soma peso + vertice anterior -> inicio
						
					++qtd_solucoes_thread;

					if (custo + mat_d(vertice[anterior(nivel)],0) < UB_local) {
						UB_local = custo + mat_d(vertice[anterior(nivel)],0);
					}
					nivel--;
				}
				//else {
					//if (custo > custoMin_d[0])
						//nivel--; //poda, LB maior que UB
				//}
			}
			else {
				vertice[nivel] = _VAZIO_;
				nivel--;
			}//else
		}//while

		sols_d[idx] = qtd_solucoes_thread;
		melhorSol_d[idx] = UB_local;

	}//dfs



}//kernel

void checkCUDAError(const char *msg) {
	hipError_t err = hipGetLastError();
	if (hipSuccess != err) {
		fprintf(stderr, "Cuda error: %s: %s.\n", msg, hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}
}

int main() {

	read();

	int *mat_d;
	int otimo_global = INFINITO;
	int qtd_sols_global = ZERO;

	upper_bound = INFINITO;

    hipEvent_t start, stop;
    HANDLE_ERROR( hipEventCreate( &start ) );
    HANDLE_ERROR( hipEventCreate( &stop ) );

    float elapsedTime;

	int nivelPreFixos = 5;//Numero de niveis prefixados; o que nos permite utilizar mais threads. 
	int nPreFixos = calculaNPrefixos(nivelPreFixos,N);

	int block_size =192; //number threads in a block
	int n_blocks = nPreFixos / block_size + (nPreFixos % block_size == 0 ? 0 : 1); // # of blocks

	int *sols_h, *sols_d;
	int *melhorSol_h, *melhorSol_d;


	short * path_h = (short*) malloc(sizeof(short) * nPreFixos * nivelPreFixos);
	short * path_d;



	sols_h = (int*)malloc(sizeof(int)*nPreFixos);
	melhorSol_h = (int*)malloc(sizeof(int)*nPreFixos);

	for(int i = 0; i<nPreFixos; ++i)		
		melhorSol_h[i] = INFINITO;
	

	
	fillFixedPaths(path_h, nivelPreFixos);




	HANDLE_ERROR( hipMalloc((void **) &mat_d, N * N * sizeof(int)));
	HANDLE_ERROR( hipMalloc((void **) &path_d, nPreFixos*nivelPreFixos*sizeof(short)));

	HANDLE_ERROR( hipMalloc((void **) &sols_d, sizeof(int)*nPreFixos));
	HANDLE_ERROR( hipMalloc((void **) &melhorSol_d, sizeof(int)*nPreFixos));

	HANDLE_ERROR( hipMemcpy(mat_d, mat_h, N * N * sizeof(int), hipMemcpyHostToDevice));
	HANDLE_ERROR( hipMemcpy(path_d, path_h, nPreFixos*nivelPreFixos*sizeof(short), hipMemcpyHostToDevice));
	HANDLE_ERROR( hipMemcpy(melhorSol_d, melhorSol_h, nPreFixos*sizeof(int), hipMemcpyHostToDevice));

	

	HANDLE_ERROR( hipDeviceSynchronize());

    HANDLE_ERROR( hipEventRecord( start, 0 ) );

	dfs_cuda_UB<<< n_blocks,block_size >>>(N,mat_d,path_d, nPreFixos , nivelPreFixos,upper_bound, sols_d,melhorSol_d);
	
	
	HANDLE_ERROR( hipDeviceSynchronize());

    HANDLE_ERROR( hipEventRecord( stop, 0 ) );
    HANDLE_ERROR( hipEventSynchronize( stop ) );


    HANDLE_ERROR( hipEventElapsedTime( &elapsedTime,start, stop ) );
    HANDLE_ERROR( hipEventDestroy( start ) );
    HANDLE_ERROR( hipEventDestroy( stop ) );

	
	HANDLE_ERROR( hipMemcpy(sols_h, sols_d, sizeof(int)*nPreFixos, hipMemcpyDeviceToHost));
	HANDLE_ERROR( hipMemcpy(melhorSol_h, melhorSol_d, sizeof(int)*nPreFixos, hipMemcpyDeviceToHost));
	
	for(int i = 0; i<nPreFixos; ++i){
		qtd_sols_global+=sols_h[i];
		if(melhorSol_h[i]<otimo_global)
			otimo_global = melhorSol_h[i];
		//printf("\nSolucoes encontradas pela thread %d: %d", i, sols_h[i]);	
		//printf("\n\tMelhor solucao encontrada pela thread %d: %d", i, melhorSol_h[i]);
	}

	puts("\n\n\n\n");
	printf("\tniveis preenchidos: %d.\n",nivelPreFixos);
	printf("\tthreads por bloco: %d.\n",block_size);
	printf("\tthreads geradas: %d.\n",nPreFixos);
	printf("\tnBlocos: %d.\n",n_blocks);
	printf("\nQuantidade de solucoes encontradas: %d.", qtd_sols_global);
	printf("\n\tOtimo global: %d,", otimo_global);
	printf( "\n\tKernel time:%3.1f ms\n", elapsedTime );
	
	HANDLE_ERROR( hipFree(mat_d));
	HANDLE_ERROR( hipFree(sols_d));
	HANDLE_ERROR( hipFree(path_d));
	HANDLE_ERROR( hipFree(melhorSol_d));

	return 0;
}
