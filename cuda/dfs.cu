#include "hip/hip_runtime.h"
// cuda_example3.cu : Defines the entry point for the console application.
//


#include <stdio.h>
#include <string.h>
#include <hip/hip_runtime.h>

#define N_h(x,y) N_h[(dimension)*(x-1)+(y-1)]
#define N_d(x,y) N_d[dimension*(x-1)+(y-1)]

#define MAX 100
#define ZERO 0
#define ONE  1
#define INICIO 1
#define TRUE 1
#define INFINITO 999999
void checkCUDAError(const char* msg);

struct nodo{

	int nivel;
	int index;
	int custo;
	int nodo_pai;
	int vflag[MAX];
};

typedef struct nodo nodo_t;

int vflag[MAX];

int N_h[] = {999999, 436, 636, 119, 131, 150, 999999, 668, 224, 305, 386, 802, 999999, 906, 31, 756, 226, 131, 999999, 602, 440, 107, 915, 275, 999999};


int dimension = 5;

// Kernel that executes on the CUDA device
__global__ void dfs(int *N_d, nodo_t *matriz_de_nodos, int *matriz_de_solucoes,int dimension ){

	/*
		@TODO: Tornar a matriz de solucoes compartilhada.
	*/

	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	nodo_t auxiliar;
	nodo_t pilha[25];
	int topo = 0;
	//int contador = 0;
	int verificados = 0;
	int posicao_solucao = idx;

	pilha[topo] = matriz_de_nodos[idx];

	/*while(topo>=ZERO){
		auxiliar = pilha[topo];
		if(auxiliar.nivel == dimension){
			matriz_de_solucoes[posicao_solucao] = auxiliar.custo;
			++posicao_solucao;
			topo--;
		}
		else{
			verificados = ZERO;
			for(int i = 1; i<=dimension; ++i){
				if(auxiliar.vflag[i] == TRUE){
					++verificados;
					continue;
				}
				else{
					auxiliar.custo+=N_d(auxiliar.index, i);
					auxiliar.index = i;
					auxiliar.nivel++;
					auxiliar.vflag[i] = TRUE;
					++topo;
					pilha[topo] = auxiliar;
					break;
				}

			}
			if(verificados == dimension){
				topo--; //desempilha
			}
		}//else
	}//while*/

}


int inline fat(int a){
	return 1;
}

nodo_t matriz[MAX];

// main routine that executes on the host
int main( void )
{


	int contador  = 0;
	//int custo = 0;
	int nivel;
	nodo_t *matriz_de_nodos_d;
	int *matriz_solucao_d;
	int *matriz_solucao_h;
	int *N_d;

	int n_blocks;
	const int block_size = 32;

	size_t size_nodos = (24)*sizeof(nodo_t);
	size_t size = (dimension)*(dimension)*(sizeof(int));
	size_t size_mat_sols = (24)*sizeof(int);


	hipMalloc( (void **)&N_d, size );
	hipMalloc( (void **)&matriz_de_nodos_d, size_nodos );
	hipMalloc( (void **)&matriz_solucao_d,size_mat_sols  );

	hipMemcpy( N_d, N_h, size, hipMemcpyHostToDevice );// passando custo para GPU

	//inicializando vflag e matriz de solucoes
	memset(vflag,ZERO, sizeof(vflag));

	matriz_solucao_h = (int *)malloc(size_mat_sols);
	for(int i = 0;i<24; ++i)
		matriz_solucao_h[i] = INFINITO;




	/*
		inicio do DFS
	*/

	/*
	*/
	vflag[INICIO] = TRUE;
	nivel = INICIO;

	for(int i = 1; i<=dimension; ++i){
		if(vflag[i] == TRUE){
			continue;}
		else{
			matriz[contador].index  = i;
			matriz[contador].custo  = N_h(INICIO,i);
			matriz[contador].nivel = nivel+1;
			matriz[contador].nodo_pai = INICIO;
			memcpy(matriz[contador].vflag, vflag, sizeof(vflag));
			matriz[contador].vflag[i] = TRUE;
			++contador;
		}
	}



	/*for(int i = 0; i<contador; ++i){
		printf("\n\t Nodo de numero %d:", i);
		printf("\nIndice: %d",matriz[i].index);
		printf("\nNivel: %d",matriz[i].nivel);
		printf("\nCusto: %d",matriz[i].custo);
		printf("\nFlag do nodo %d: ",i);
		for(int j = 1; j<=dimension; ++j){
			printf("%d",matriz[i].vflag[j]);
		}
	}

	node = matriz[0];

	printf("\n\nNodo dps da copia:");


	printf("\nIndice: %d",node.index);
	printf("\nNivel: %d",node.nivel);
	printf("\nCusto: %d",node.custo);
	printf("\nFlag do nodo:");
	for(int j = 1; j<=dimension; ++j){
		printf("%d",node.vflag[j]);
	}*/


	hipMemcpy(matriz_de_nodos_d, matriz, contador*(sizeof(nodo_t)), hipMemcpyHostToDevice );
    checkCUDAError("memcpy1");

	hipMemcpy(matriz_solucao_d, matriz_solucao_h, size_mat_sols, hipMemcpyHostToDevice );
    checkCUDAError("memcpy2");

	n_blocks   = contador / block_size + ( contador % block_size == 0 ? 0 : 1 );

	hipDeviceSynchronize();
	dfs<<<n_blocks, block_size>>>(N_d, matriz_de_nodos_d, matriz_solucao_d, dimension);
	checkCUDAError("kernel invocation");
    hipDeviceSynchronize();
	hipMemcpy( matriz_solucao_h, matriz_solucao_d, size_mat_sols, hipMemcpyDeviceToHost );
 	checkCUDAError("memcpy-d-h");

	puts("\nSolucoes");
	for(int i = 0; i<24; ++i){
		printf("\nSulcao %d: %d", i, matriz_solucao_h[i]);
	}

	hipFree( N_d );
 	/*
		colocar o free das outras variaveis
	*/

   return 0;
}

void checkCUDAError(const char *msg)
{
    hipError_t err = hipGetLastError();
    if( hipSuccess != err)
    {
        fprintf(stderr, "Cuda error: %s: %s.\n", msg,
                             hipGetErrorString( err) );
        exit(EXIT_FAILURE);
    }
}
