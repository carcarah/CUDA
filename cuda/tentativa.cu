// cuda_example3.cu : Defines the entry point for the console application.
//


#include <stdio.h>
#include <string.h>
#include <hip/hip_runtime.h>

const int N = 64;

__global__ void foo( float **a, int N )
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    if ( i < N && j<N )
        a[i][j] = 1;
}


// main routine that executes on the host
int main( void )
{
    const int block_size = 4;
    int n_blocks;
    
    dim3 dimblock(block_size, block_size);

   
    float a_h[N][N], **a_d; // Pointer to host & device arrays
   


    size_t size = N * N * sizeof( float );

    hipMalloc( (void **)&a_d, size ); // Allocate array on device
    

    // Initialize host array and copy it to CUDA device
    for ( int i = 0; i < N; i++){
        for(int j = 0; j<N; j++){
	    a_h[i][j] = (float)i;
        }
     }


    for ( int i = 0; i < N; i++ ){
	
        for(int j = 0; j<N; j++){
	    printf("%d ",(int)a_h[i][j]);
        }
    }
    puts("\n");

   
   /*
       invocando o kernel
   */
   n_blocks   = N / block_size + ( N % block_size == 0 ? 0 : 1 );
   hipMemcpy( a_d, a_h, size, hipMemcpyHostToDevice );

   foo<<< n_blocks, dimblock >>> ( a_d, N );

   hipMemcpy( a_h, a_d,size, hipMemcpyDeviceToHost );

   puts("\nDEVICE - HOST:\n");

   for ( int i = 0; i < N; i++ ){
	
       for(int j = 0; j<N; j++){
	    printf("%d ",(int)a_h[i][j]);
        }
    }
    puts("\n");

    //free( a_h );

    hipFree( a_d );
    return 0;
}
