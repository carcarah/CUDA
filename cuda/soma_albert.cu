// cuda_example3.cu : Defines the entry point for the console application.
//


#include <stdio.h>
#include <hip/hip_runtime.h>
#include <string.h>


// Kernel that executes on the CUDA device
__global__ void square_array( float *a, long int N, long int b )
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if ( ((2*b)*idx+b) < N){
        a[idx*2*b] = a[idx*2*b]+a[((2*b)*idx+b)];
    	 a[((2*b)*idx+b)] = 0;
       
    }
}

typedef struct hipDeviceProp_t cudaDevProp_t;

// main routine that executes on the host
int main( void )
{
    
    int ct,dev;
    long int b = 1;
    int passos = 0;
    cudaDevProp_t prop;

    float *a_h, *a_d; // Pointer to host & device arrays
    const long int N = 30000; // Number of elements in arrays

    size_t size = N * sizeof( float );

    hipGetDeviceCount(&ct); //is there a cuda device??

    if(ct == 0){ 
        printf("\nNo CUDA device found.\n");
        exit(0);
    }

    a_h = (float *)malloc( size );    // Allocate array on host
    hipMalloc( (void **)&a_d, size ); // Allocate array on device

    // Initialize host array and copy it to CUDA device
    for ( long int i = 0; i < N; i++ )
        a_h[i] = (float)1;

    hipMemcpy( a_d, a_h, size, hipMemcpyHostToDevice );

    // kernel initialization
    hipGetDevice(&dev);
    hipGetDeviceProperties(&prop,dev);
    int block_size = prop.maxThreadsPerBlock; //ATENTION
    int n_blocks   = N / block_size + ( N % block_size == 0 ? 0 : 1 );

    while(b<N){
        passos++;
    	square_array <<< n_blocks, block_size >>> ( a_d, N,b );
        b = b*2;
        hipMemcpy( a_h, a_d, sizeof( float ) * N, hipMemcpyDeviceToHost );
        puts("\n\n");
    // Print results
      // for ( int i = 0; i < N; i++ )
          printf( " %f\n",a_h[0] ); // Cleanup
    }
    // Retrieve result from device and store it in host array

    printf("\n\tPASSOS: %d", passos);
    free( a_h );
    hipFree( a_d );
    return 0;
}
