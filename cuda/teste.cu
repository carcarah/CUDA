#include "hip/hip_runtime.h"
hipArray* cu_array;
texture<float, 2> tex;

// Allocate array
hipMallocArray(&cu_array, hipCreateChannelDesc<float>(), width, height);

// Copy image data to array
hipMemcpy(cu_array, image, width*height, hipMemcpyHostToDevice);

// Bind the array to the texture
hipBindTexture(tex, cu_array);

// Run kernel
dim3 blockDim(16, 16, 1);
dim3 gridDim(width / blockDim.x, height / blockDim.y, 1);
kernel<<< gridDim, blockDim, 0 >>>(d_odata, width, height);
hipUnbindTexture(tex);

__global__ void kernel(float* odata, int height, int width)
{
unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
unsigned int y = blockIdx.y*blockDim.y + threadIdx.y;
float c = texfetch(tex, x, y);
odata[y*width+x] = c;
}
