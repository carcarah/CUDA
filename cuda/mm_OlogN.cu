#include "hip/hip_runtime.h"
#include <stdio.h>
#include <string.h>
#include <cstdlib>
#include <cmath>
#include <omp.h>
#include <ctime>
#include <hip/hip_runtime.h>

#define M 10
#define N 10
#define T 1500

#define A_h(i,j) A_h[T*i+j]
#define B_h(i,j) B_h[M*i+j]
#define C_h(i,j) C_h[M*i+j]

#define A_d(i,j) A_d[T*i+j]
#define B_d(i,j) B_d[M*i+j]

#define C_dd(i,j,k) C_dd[M*N*k+M*i+j]

#define C_dh(i,j,k) C_dh[M*N*k+M*i+j]

void checkCUDAError(const char* msg);
//int A_h[] = {1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1};

//int A_h[] = {1,0,0,0,0,1,0,0,0,0,1,0,0,0,0,1};

//int B_h[] = {10,10,10,10,10,10,10,10,10,10,10,10,10,10,10,10};

int A_h[N*T];
int B_h[T*M];


__global__ void bar(int *A_d,int *B_d, int *C_dd, int total){
	int i,j,k;
	int b = 1;
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	if(idx < total){
		k=idx/(N*M);
		j=(idx-(N*M*k))%N;
		i=(idx-(N*M*k))/N;
		C_dd(i,j,k) = A_d(i,k)*B_d(k,j);
		while((2*b)*k+b < T){
			//__syncthreads(); errado
			C_dd(i,j,(k*2*b)) = C_dd(i,j,(k*2*b))+C_dd(i,j,((2*b)*k+b));
    	 		b*=2;
		}
	}

}



__global__ void soma(int *C_dd, int b, int total){

	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	int i,j,k;

	if(idx<total){
		k=idx/(N*M);
		j=(idx-(N*M*k))%M;
		i=(idx-(N*M*k))/M;
		if ( ((2*b)*k+b) < T){
        		C_dd(i,j,(k*2*b)) = C_dd(i,j,(k*2*b))+C_dd(i,j,((2*b)*k+b));
    	 		//C_dd(i,j,(2*b)*k+b)) = 0;

    		}
	}

}
int main(){

    /*int A_h[M*M];
    int B_h[M*M];
    int C_h[M*M];*/

    int *A_d, *B_d, *C_d;
    int *C_dd;
    int *C_dh;

    int qtd_lin_a,qtd_lin_b,qtd_col_a,qtd_col_b;



    int block_size;
    int n_blocks;

    int b =1;
    int cont = 0;

    size_t size = M * M *sizeof( int );
    size_t sizeCdd = M*N*T*sizeof(int);

    srand(time(NULL));

    for(int i = 0; i<N; ++i){
        for(int j = 0; j<T; ++j){
            A_h(i,j) =rand() % 10;
        }
    }

    for(int i = 0; i<T; ++i){
        for(int j = 0; j<M; ++j){
            B_h(i,j) =rand() % 10;
        }
    }


/*
    qtd_lin_a = M;
    qtd_col_a = N;

    qtd_lin_b = qtd_col_a;
    qtd_col_b =;
*/


   /* puts("\n\nMultiplicacao de matrizes");
    puts("\n\tDigite a qtd de linhas de A: ");
    scanf("%d", &qtd_lin_a);
    puts("\n\tDigite a qtd de colunas de A: ");
    scanf("%d", &qtd_col_a);
    qtd_lin_b = qtd_col_a;
    puts("\n\tDigite a qtd de colunas de B: ");
    scanf("%d", &qtd_col_b);*/

/*
    srand ();

    for(int i = 0; i<qtd_lin_a; ++i){
        for(int j = 0; j<qtd_col_a; ++j){
            A_h(i,j) =rand() % 10;
        }
    }

    for(int i = 0; i<qtd_lin_b; ++i){
        for(int j = 0; j<qtd_col_b; ++j){
            B_h(i,j) =rand() % 10;
        }
    }
*/
     C_dh = (int *)malloc( sizeCdd );
    hipMalloc( (void **)&A_d, size );
    hipMemcpy( A_d, A_h, size, hipMemcpyHostToDevice );

    hipMalloc( (void **)&B_d, size );
    hipMemcpy( B_d, B_h, size, hipMemcpyHostToDevice );

//    hipMalloc( (void **)&C_d, size );
    hipMalloc( (void **)&C_dd, sizeCdd);



    /***************/
    /**KERNEL******/
    /*block_size = 32;
    n_blocks   = M / block_size + ( M % block_size == 0 ? 0 : 1 );
    foo<<< n_blocks, block_size >>> (A_d, B_d, C_d,qtd_col_a,qtd_col_a,M);
    hipMemcpy( C_h, C_d, sizeof( int ) * M * M, hipMemcpyDeviceToHost );*/
    hipDeviceSynchronize();


    block_size = 64;
    n_blocks   = (M*N*T) / block_size + ( (M*N*T) % block_size == 0 ? 0 : 1 );
    bar<<<n_blocks, block_size>>>(A_d, B_d, C_dd, (M*N*T));
    checkCUDAError("kernel invocation");
    ++cont;

    hipMemcpy( C_dh, C_dd, sizeof( int ) *M*N, hipMemcpyDeviceToHost );

   /* for(int i = 0; i<N; ++i){
	    for(int j = 0; j<M; ++j){
			printf(" %d", C_dh(i,j,0));

		}
		puts("\n");
	}
    printf("\n\tContador: %d", cont);	*/
    return 0;
}

void checkCUDAError(const char *msg)
{
    hipError_t err = hipGetLastError();
    if( hipSuccess != err)
    {
        fprintf(stderr, "Cuda error: %s: %s.\n", msg,
                             hipGetErrorString( err) );
        exit(EXIT_FAILURE);
    }
}
