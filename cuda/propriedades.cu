#include "hip/hip_runtime.h"
#include<stdio.h>
#include <hip/hip_runtime.h>
#include <string.h>


typedef struct hipDeviceProp_t cudaDevProp_t;

int main(int argc, char** argv){
    int ct,dev;
    cudaDevProp_t prop;
 
    hipGetDeviceCount(&ct); //Verificar se existe dispositivo cuda. Passa o endereco de uma variavel inteira.

    if(ct == 0){
        printf("\nNo CUDA device found.\n");
        exit(0);
    }
    else{euahieaiueuiaeaihea
	hipGetDevice(&dev); /*verificara qual dos dispositivos esta ativo*/

	/*
	    Se existirem multiplos dispositivos, funcao hipSetDevice pode ser utilizada
	    
	*/
        hipGetDeviceProperties(&prop,dev);
        
	printf("\n%d Device Found\n",ct);
        printf("\nThe Device ID is %d\n",dev);
	printf("\tDevice Name : %s",prop.name);
        printf("\nThe Properties of the Device with ID %d are:\n",dev);
	printf("\n\tCompute capability: %d.%d", prop.major, prop.minor);
	printf("\n\tCompute mode: %d", prop.computeMode);
	printf("\n\tNumber of multiprocessors: %d", prop.multiProcessorCount);	
	printf("\n\tDevice clock rate (Khz): %d", prop.clockRate);
	
	printf("\n\tDevice Memory Size (in Mbytes) : %lu",(unsigned long)prop.totalGlobalMem/1000000);
	printf("\n\tShared memory per block (in bytes): %lu",(unsigned long)prop.sharedMemPerBlock);
	printf("\n\tNumber of registers per block: %d", prop.regsPerBlock);
	puts("\n");
	printf("\n\tNumber of threads per warp: %d", prop.warpSize);
	printf("\n\tNumber of threads per block: %d", prop.maxThreadsPerBlock);
	printf("\n\tMax dimensions of a block:\n\t\tX:%d, Y:%d, Z:%d", prop.maxThreadsDim[0], prop.maxThreadsDim[1], prop.maxThreadsDim[2]);
	printf("\n\tMax dimensions of a grid:\n\t\tX:%d, Y:%d, Z:%d", prop.maxGridSize[0], prop.maxGridSize[1], prop.maxGridSize[2]);
	
    }

        
        /*
	char name[256]; *
        size_t totalGlobalMem; *
        size_t sharedMemPerBlock; *
        int regsPerBlock; *
        int warpSize; *
        size_t memPitch;
        int maxThreadsPerBlock;*
        int maxThreadsDim[3];*
        int maxGridSize[3];*

        size_t totalConstMem;
        int major;*
        int minor;*
        int clockRate;*
        size_t textureAlignment;
        int deviceOverlap;
        int multiProcessorCount*;
        int kernelExecTimeoutEnabled;
        int integrated;
        int canMapHostMemory;
        int computeMode;*/

    printf("\n");

    return 0;
}

