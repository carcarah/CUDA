#include "hip/hip_runtime.h"
// cuda_example3.cu : Defines the entry point for the console application.
#include <stdio.h>
#include <hip/hip_runtime.h>

// Kernel that executes on the CUDA device

__global__ void add_two_vectors(float *a, float *b, float *c, int N)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if ( idx < N ){ 
		c[idx] = a[idx] + b[idx];
    }
}




// main routine that executes on the host
int main( void )
{
    float *a_h, *a_d; // Pointer to host & device arrays
    float *b_h, *b_d;
    float *c_h, *c_d;

    const int N = 10000000; // Number of elements in arrays
    int block_size = 4; //number threads in a block
    int n_blocks   = N / block_size + ( N % block_size == 0 ? 0 : 1 ); // # of blocks
	.
	.
	.
	
		
    size_t size = N * sizeof( float ); //for alocation. aloca N vezes o tamanho do tipo

    a_h = (float *)malloc( size );    // Allocate array on host
    b_h = (float *)malloc( size );
    c_h = (float *)malloc( size );

    hipMalloc( (void **)&a_d, size ); // Allocate array on device
    hipMalloc( (void **)&b_d, size );
    hipMalloc( (void **)&c_d, size );

    // Initialize host array and copy it to CUDA device
    for ( int i = 0; i < N; i++ )
        a_h[i] = (float)i; //float pois gpu trabalha com FPO
    hipMemcpy( a_d, a_h, size, hipMemcpyHostToDevice ); //mesma coisa do memcopy, sendo que copiando um host inicializado pro device(gpu)
				                  //ponteiros para os vetores, size, operacao

    for ( int i = 0; i < N; i++ )
        b_h[i] = (float)i;
    hipMemcpy( b_d, b_h, size, hipMemcpyHostToDevice );

    

    //square_array <<< n_blocks, block_size >>> ( a_d, N ); //chamando o kernel com o vertor iniciado na GPU
    
    add_two_vectors <<< n_blocks,block_size >>> (a_d,b_d,c_d,N); 
	do_some_host_computation( );
	hipMemcpy(...);
    
// Retrieve result from device and store it in host array
    hipMemcpy( c_h, c_d, sizeof( float ) * N, hipMemcpyDeviceToHost ); //agora a operacao contra'ria a linha 38 esta sendo realizada.

   // Print results
    for ( int i = 0; i < N; i++ )
        printf( "%d %f\n", i, c_h[i] ); // Cleanup
    
    free( a_h );
    free( b_h );
    free( c_h );
    
    hipFree( a_d );
    hipFree( b_d );
    hipFree( c_d );
    return 0;
}
