#include "hip/hip_runtime.h"
// cuda_example3.cu : Defines the entry point for the console application.
//


#include <stdio.h>
#include <hip/hip_runtime.h>
#include <string.h>

#define A(x,y) A[M*x+y]
#define a_h(x,y) a_h[M*x+y]

typedef struct hipDeviceProp_t cudaDevProp_t;



// Kernel that executes on the CUDA device

//A[i][j] = ordem do elemento na matriz, de 0 -> n²-1
__global__ void bar(float *A, int N, int M){
	int i = blockIdx.x * blockDim.x + threadIdx.x; //blockIdx*blockDim e blockIdx*blockDim garante que toda a matriz seja coberda por 													   //threads
	int j = blockIdx.y * blockDim.y + threadIdx.y;
	
	if(i<N && j<M){
		A(i,j) = M*i+j;}
}

void checkCUDAError(const char *msg)
{
    hipError_t err = hipGetLastError(); //erro da ultima opereacao cuda chamada
    if( hipSuccess != err) 
    {
        fprintf(stderr, "Cuda error: %s: %s.\n", msg, 
                                  hipGetErrorString( err) );
        exit(EXIT_FAILURE);
    }                         
}

// main routine that executes on the host
int main( void )
{
    float *a_h, *a_d; 
    const int N = 1002; 
    const int M = 1002;

	int ct, dev;
	cudaDevProp_t prop;
	

	hipGetDeviceCount(&ct);
    if(ct == 0){ 
        printf("\nNo CUDA device found.\n");
        exit(0);
    }
    hipGetDevice(&dev);
    hipGetDeviceProperties(&prop,dev);

	dim3 bDim(22,22); //threads por bloco. Nao podem ultrapassar a capacidade da VGA
						//se o Z nao for definido, fica Z=1

  	dim3 gDim((N/bDim.x)+( N % bDim.x == 0 ? 0 : 1 ),M/bDim.y+( M % bDim.y == 0 ? 0 : 1 )); //~numBlocks

    size_t size = N * M *sizeof( float );
  
    a_h = (float *)malloc( size );   //Tudo sera alocado da mesma forma, pois temos matriz A[N*M] mas estamos visualizando A[N][M]
  		
    hipMalloc( (void **)&a_d, size ); 
    hipMemcpy( a_d, a_h, size, hipMemcpyHostToDevice );

	checkCUDAError("memcpy"); //caro tenha ocorrido erro, retorna o tipo e em qual operacao ocorreu.
	
	bar<<<gDim, bDim>>>(a_d, N,M);

	hipDeviceSynchronize(); // bloqueia o device ate que a execucao do kernel tenha sido concluida. Retorna erro ou sucesso.

    checkCUDAError("kernel invocation");

	/*
		Atencao: sem cudaThreadSync, o programa retornaria os erros do memcopy, nao do kernel.
	*/

    hipMemcpy( a_h, a_d, sizeof( float ) * N * M, hipMemcpyDeviceToHost ); //recuperando resultados
    checkCUDAError("memcpy"); //checa erro ao recuperar os resultados


   /*for ( int i = 0; i < N; i++ ){
		printf("%d[ ", i);
		for(int j = 0; j<M; ++j){
    		printf( "%d ",(int)a_h(i,j) ); 
    	}
		puts(" ]");
		puts("\n");
    }*/
	
	printf("\n\n%d\n\n", (int)a_h((N-1),(M-1))); 
	if((int)a_h((N-1),(M-1))!=(N*N-1)){
		printf("\n\nKernel com erros de programacao");
		exit(1);
	}
    free( a_h );
    hipFree( a_d );

	printf("\n\tPrograma sem erros CUDA...\n\n"); //se o programa chegou aqui, ele nao apresenta erros.
    return 0;
}
