#include <stdio.h>
#include <string.h>
#include <cstdlib>
#include <cmath>
#include <omp.h>
#include <ctime>
#include <hip/hip_runtime.h>



#define A_h(x,y) A_h[M*x+y]
#define B_h(x,y) B_h[M*x+y]
#define C_h(x,y) C_h[M*x+y]

#define A_d(x,y) A_d[M*x+y]
#define B_d(x,y) B_d[M*x+y]
#define C_d(x,y) C_d[M*x+y]

__global__ void foo(int *A_d,int *B_d, int *C_d, int qtd_col_a, int qtd_col_b,int M ){
    int acumulador;
	int linha = blockIdx.x * blockDim.x + threadIdx.x; //blockIdx*blockDim e blockIdx*blockDim garante que toda a matriz seja coberda por 													   //threads

	if(linha<M){
        for(int coluna = 0; coluna< qtd_col_b; ++coluna){
            acumulador = 0;
            for(int i = 0; i<qtd_col_a; ++i){
                acumulador = acumulador + (A_d(linha,i)*B_d(i,coluna));
            }
            C_d(linha,coluna) = acumulador;
        }
    }
}

int main(){
    int M = 500;
    int A_h[M*M];
    int B_h[M*M];
    int C_h[M*M];

    int *A_d, *B_d, *C_d;

    int qtd_lin_a,qtd_lin_b,qtd_col_a,qtd_col_b;



    int block_size;
    int n_blocks;




    size_t size = M * M *sizeof( int );




    qtd_lin_a = M;
    qtd_col_a = M;

    qtd_lin_b = qtd_col_a;
    qtd_col_b =M;



   /* puts("\n\nMultiplicacao de matrizes");
    puts("\n\tDigite a qtd de linhas de A: ");
    scanf("%d", &qtd_lin_a);
    puts("\n\tDigite a qtd de colunas de A: ");
    scanf("%d", &qtd_col_a);
    qtd_lin_b = qtd_col_a;
    puts("\n\tDigite a qtd de colunas de B: ");
    scanf("%d", &qtd_col_b);*/


    srand(time(NULL) );

    for(int i = 0; i<qtd_lin_a; ++i){
        for(int j = 0; j<qtd_col_a; ++j){
            A_h(i,j) =rand() % 10;
        }
    }

    for(int i = 0; i<qtd_lin_b; ++i){
        for(int j = 0; j<qtd_col_b; ++j){
            B_h(i,j) =rand() % 10;
        }
    }


    hipMalloc( (void **)&A_d, size );
    hipMemcpy( A_d, A_h, size, hipMemcpyHostToDevice );

    hipMalloc( (void **)&B_d, size );
    hipMemcpy( B_d, B_h, size, hipMemcpyHostToDevice );

    hipMalloc( (void **)&C_d, size );



    /***************/
    /**KERNEL******/
    block_size = 32;
    n_blocks   = M / block_size + ( M % block_size == 0 ? 0 : 1 );
    foo<<< n_blocks, block_size >>> (A_d, B_d, C_d,qtd_col_a,qtd_col_a,M);
    hipMemcpy( C_h, C_d, sizeof( int ) * M * M, hipMemcpyDeviceToHost );
    return 0;
}


