#include <stdio.h>
#include <string.h>
#include <cstdlib>
#include <cmath>
#include <omp.h>
#include <ctime>
#include <hip/hip_runtime.h>

#define M 500
#define N 500
#define T 500

#define A_h(i,j) A_h[M*i+j]
#define B_h(i,j) B_h[M*i+j]
#define C_h(i,j) C_h[M*i+j]

#define A_d(i,j) A_d[M*i+j]
#define B_d(i,j) B_d[M*i+j]

#define C_dd(i,j) C_dd[M*i+j]

#define C_dh(i,j) C_dh[M*i+j]


//int A_h[] = {1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1};

//int A_h[] = {1,0,0,0,0,1,0,0,0,0,1,0,0,0,0,1};

//int B_h[] = {10,10,10,10,10,10,10,10,10,10,10,10,10,10,10,10};

int A_h[M*T];
int B_h[M*T];


__global__ void bar(int *A_d,int *B_d, int *C_dd, int total){
	int i,j,k;

	int idx = blockIdx.x * blockDim.x + threadIdx.x; 
	
	if(idx < total){
		
		j=(idx)%M;
		i=(idx)/M;
		C_dd(i,j) = 0;
		for(k = 0; k<T; ++k){
			C_dd(i,j) += A_d(i,k)*B_d(k,j);
                }	
	}
	
}


int main(){
    
    /*int A_h[M*M];
    int B_h[M*M];
    int C_h[M*M];*/

    int *A_d, *B_d, *C_d;
    int *C_dd;
    int *C_dh;

    int qtd_lin_a,qtd_lin_b,qtd_col_a,qtd_col_b;



    int block_size;
    int n_blocks;

    int b =1;
    int cont = 0;

    size_t size = M * M *sizeof( int );
    size_t sizeCdd = M*N*sizeof(int);

    srand(time(NULL));

    for(int i = 0; i<M; ++i){
        for(int j = 0; j<N; ++j){
            A_h(i,j) =rand() % 10;
        }
    }

    for(int i = 0; i<M; ++i){
        for(int j = 0; j<N; ++j){
            B_h(i,j) =rand() % 10;
        }
    }


/*
    qtd_lin_a = M;
    qtd_col_a = N;

    qtd_lin_b = qtd_col_a;
    qtd_col_b =;
*/


   /* puts("\n\nMultiplicacao de matrizes");
    puts("\n\tDigite a qtd de linhas de A: ");
    scanf("%d", &qtd_lin_a);
    puts("\n\tDigite a qtd de colunas de A: ");
    scanf("%d", &qtd_col_a);
    qtd_lin_b = qtd_col_a;
    puts("\n\tDigite a qtd de colunas de B: ");
    scanf("%d", &qtd_col_b);*/

/*
    srand ();

    for(int i = 0; i<qtd_lin_a; ++i){
        for(int j = 0; j<qtd_col_a; ++j){
            A_h(i,j) =rand() % 10;
        }
    }

    for(int i = 0; i<qtd_lin_b; ++i){
        for(int j = 0; j<qtd_col_b; ++j){
            B_h(i,j) =rand() % 10;
        }
    }
*/
     C_dh = (int *)malloc( sizeCdd );   
    hipMalloc( (void **)&A_d, size );
    hipMemcpy( A_d, A_h, size, hipMemcpyHostToDevice );

    hipMalloc( (void **)&B_d, size );
    hipMemcpy( B_d, B_h, size, hipMemcpyHostToDevice );

    hipMalloc( (void **)&C_d, size );
    hipMalloc( (void **)&C_dd, sizeCdd);



    /***************/
    /**KERNEL******/
    /*block_size = 32;
    n_blocks   = M / block_size + ( M % block_size == 0 ? 0 : 1 );
    foo<<< n_blocks, block_size >>> (A_d, B_d, C_d,qtd_col_a,qtd_col_a,M);
    cudaMemcpy( C_h, C_d, sizeof( int ) * M * M, cudaMemcpyDeviceToHost );*/
	
    block_size = 32;
    n_blocks   = (M*N) / block_size + ( (M*N) % block_size == 0 ? 0 : 1 );
    bar<<<n_blocks, block_size>>>(A_d, B_d, C_dd, (M*N));
    //++cont;     
	// cudaMemcpy( C_dh, C_dd, sizeof( int ) *M*N*T, cudaMemcpyDeviceToHost );

   /* for(int k = 0; k<T; ++k){
	for(int i = 0; i<N; ++i){
	    for(int j = 0; j<M; ++j){
			printf(" %d", C_dh(i,j,k));
			
		}
		puts("\n");	
	}
	puts("---\n");
    }
*/
    hipMemcpy( C_dh, C_dd, sizeof( int ) *M*N, hipMemcpyDeviceToHost );

    /*for(int i = 0; i<N; ++i){
	    for(int j = 0; j<M; ++j){
			printf(" %d", C_dh(i,j,0));
			
		}
		puts("\n");	
	}*/
    printf("\n\tContador: %d", cont);	
    return 0;
}


