// cuda_example3.cu : Defines the entry point for the console application.
//


#include <stdio.h>
#include <hip/hip_runtime.h>
#include <string.h>

#define A(x,y) A[M*x+y]
#define a_h(x,y) a_h[M*x+y]
typedef struct hipDeviceProp_t cudaDevProp_t;



// Kernel that executes on the CUDA device
__global__ void foo( float *A, int N, int M)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if ( idx < N ){ //aqui idx representa a linha. Cada elemento (idx,y) e gerado
		for(int i = 0; i<M; ++i){
			A(idx,i) = M*idx+i;
		}
    }
  
}



// main routine that executes on the host
int main( void )
{
    float *a_h, *a_d; 
    const int N = 1000; 
    const int M = 10000;
	int ct, dev;
	cudaDevProp_t prop;

	hipGetDeviceCount(&ct); //is there a cuda device??

    if(ct == 0){ 
        printf("\nNo CUDA device found.\n");
        exit(0);
    }
	
    hipGetDevice(&dev);
    hipGetDeviceProperties(&prop,dev);

    int block_size = prop.maxThreadsPerBlock; //maior quantidade de threads permitida em um bloco unidimensional
    int n_blocks   = N*M / block_size + ( N % block_size == 0 ? 0 : 1 );

    size_t size = N * M *sizeof( float );

    a_h = (float *)malloc( size );    //Tudo sera alocado da mesma forma, pois temos matriz A[N*M] mas estamos visualizando A[N][M]
    hipMalloc( (void **)&a_d, size ); 
    hipMemcpy( a_d, a_h, size, hipMemcpyHostToDevice );

  

    foo <<< n_blocks, block_size >>> ( a_d, N,M );


    hipMemcpy( a_h, a_d, sizeof( float ) * N * M, hipMemcpyDeviceToHost ); //recuperando resultados
    
   /*for ( int i = 0; i < N; i++ ){
		printf("%d[ ", i);
		for(int j = 0; j<M; ++j){
    		printf( "%d ",(int)a_h(i,j) ); 
    	}
		puts(" ]");
		puts("\n");
    }*/
	printf("\n\n%d\n\n", (int)a_h((N-1),(M-1))); 

    free( a_h );
    hipFree( a_d );
    return 0;
}
